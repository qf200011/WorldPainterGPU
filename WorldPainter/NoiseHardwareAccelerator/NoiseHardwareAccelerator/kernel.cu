#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "org_pepsoft_worldpainter_exporting_NoiseHardwareAccelerator.h"
#include <jni.h>
#include <stdlib.h>
#include "NoiseGeneration.h"
#include <windows.h>

#include <chrono>
#include <stdio.h>
#include <ctime>


#define TINY_BLOBS 4.099f
#define TILE_SIZE 128
#define REGION_SIZE 512
#define CHUNK_SIZE 16
#define TILES_PER_REGION_AXIS 5
#define CHUNKS_PER_TILE_AXIS 8
#define MAX_HEIGHT 128
#define MIN_HEIGHT -64
#define X_ARRAY_SIZE 680
#define Y_ARRAY_SIZE 680

hipError_t noiseWithCuda(int* p, float* chances, float* regionArrayX, float* regionArrayY, float* regionArrayZ, byte* output, int totalHeight, long long& dev_regionArrayXPtr, long long& dev_regionArrayYPtr, long long& dev_regionArrayZPtr, long long& dev_pPtr, long long& dev_outputPtr);
void getRegionArray(float* regionArrayX, float* regionArrayY, float* regionArrayZ, int minHeight, int maxHeight, int regionX, int regionY);
void getPArray(int* p, JNIEnv* env, jlong seed);
void swap(int* array, int index1, int index2);


__global__ void generateNoise(int* p, float* chances, float* regionArrayX, float* regionArrayY, float* regionArrayZ, byte *output, int totalHeight)
{
    /*int x=128;
    int y=128;
    int z=64+61;
    printf("P is %d\n", p[131]);
    output[x+ y*Y_ARRAY_SIZE + z* Z_ARRAY_SIZE]= getPerlinNoiseAt(regionArrayX[x], regionArrayY[y], regionArrayZ[z], p);*/

    /*if (blockIdx.x == 128 && blockIdx.y == 128 && threadIdx.x == 125) {
        printf("Special value: %f in spot %d\n", getPerlinNoiseAt(regionArrayX[blockIdx.x], regionArrayY[blockIdx.y], regionArrayZ[threadIdx.x], p), blockIdx.x + (blockIdx.y * X_ARRAY_SIZE) + (threadIdx.x * X_ARRAY_SIZE * Y_ARRAY_SIZE));
    }

    if (blockIdx.x + (blockIdx.y * X_ARRAY_SIZE) + (threadIdx.x * X_ARRAY_SIZE * Y_ARRAY_SIZE) == 122432) {
        printf("I hit the index with X:%d Y:%d Z:%d\n", blockIdx.x, blockIdx.y, threadIdx.x);
        printf("(%d + %d  * %d ) + (%d  * %d )", blockIdx.x, blockIdx.y, Y_ARRAY_SIZE, threadIdx.x, Z_ARRAY_SIZE);
    }*/

    float outputNoise = getPerlinNoiseAt(regionArrayX[blockIdx.x], regionArrayY[blockIdx.y], regionArrayZ[threadIdx.x], p);


    bool shouldSetMaterial = outputNoise >= chances[8];



    int index = blockIdx.x + (blockIdx.y * X_ARRAY_SIZE) + (threadIdx.x * X_ARRAY_SIZE * totalHeight);
    int byteIndex = index / 8;
    int bitIndex = index % 8;

    if (shouldSetMaterial && bitIndex == 0) {
        output[byteIndex] |= (1 << bitIndex);
    }
    __syncthreads();
    if (shouldSetMaterial && bitIndex == 1) {
        output[byteIndex] |= (1 << bitIndex);
       
    }
    __syncthreads();
    if (shouldSetMaterial && bitIndex == 2) {
        output[byteIndex] |= (1 << bitIndex);
        
    }
    __syncthreads();
    if (shouldSetMaterial && bitIndex == 3) {
        output[byteIndex] |= (1 << bitIndex);
        
    }
    __syncthreads();
    if (shouldSetMaterial && bitIndex == 4) {
        output[byteIndex] |= (1 << bitIndex);
        
    }
    __syncthreads();
    if (shouldSetMaterial && bitIndex == 5) {
        output[byteIndex] |= (1 << bitIndex);
        
    }
    __syncthreads();
    if (shouldSetMaterial && bitIndex == 6) {
        output[byteIndex] |= (1 << bitIndex);
        
    }
    __syncthreads();
    if (shouldSetMaterial && bitIndex == 7) {
        output[byteIndex] |= (1 << bitIndex);
        output[blockIdx.x + (blockIdx.y * X_ARRAY_SIZE) + (threadIdx.x * X_ARRAY_SIZE * totalHeight)];
    }

}

int main()
{
    //float regionArrayX[X_ARRAY_SIZE]; //dx
    //float regionArrayY[Y_ARRAY_SIZE]; //dy
    ////float regionArrayZ[Z_ARRAY_SIZE]; //dz but shifted
    //int p[512];
    ////float output[X_ARRAY_SIZE * Y_ARRAY_SIZE * Z_ARRAY_SIZE];
    ////[TILE_SIZE * TILES_PER_REGION_AXIS] [TILE_SIZE * TILES_PER_REGION_AXIS] [totalHeight]
    ////getRegionArray(regionArrayX, regionArrayY, regionArrayZ, Z_ARRAY_SIZE, 0, 0);
    ////getPArray(p);
 
    ///*for (int i = 0; i < 512; i++) {
    //    printf("%d: %d\n", i, p[i]);
    //}*/

    //// Add vectors in parallel.

    ////hipError_t cudaStatus = noiseWithCuda(p, regionArrayX, regionArrayY, regionArrayZ, output);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addWithCuda failed!");
    //    delete[] regionArrayX;
    //    delete[] regionArrayY;
    //    delete[] regionArrayZ;
    //    delete[] output;
    //    delete[] p;
    //    return 1;
    //}

    //delete[] regionArrayX;
    //delete[] regionArrayY;
    //delete[] regionArrayZ;
    //delete[] output;
    //delete[] p;
    ///*for (int i= 0; i < X_ARRAY_SIZE * Y_ARRAY_SIZE * Z_ARRAY_SIZE; i++) {
    //    printf("%d: %.6f\n", i, output[i]);
    //}*/

    //// hipDeviceReset must be called before exiting in order for profiling and
    //// tracing tools such as Nsight and Visual Profiler to show complete traces.
    //cudaStatus = hipDeviceReset();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceReset failed!");
    //    return 1;
    //}

    //return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t noiseWithCuda(int* p, float* chances, float* regionArrayX, float* regionArrayY, float* regionArrayZ, byte*  output,int totalHeight, long long &dev_regionArrayXPtr, long long &dev_regionArrayYPtr, long long& dev_regionArrayZPtr, long long& dev_pPtr, long long &dev_outputPtr)
{
    std::clock_t c_start = std::clock();

    int* dev_p;
    float* dev_regionArrayX;
    float* dev_regionArrayY;
    float* dev_regionArrayZ;
    byte* dev_output;
    float* dev_chances;

    hipError_t cudaStatus;
    auto t_start = std::chrono::high_resolution_clock::now();
    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    if (dev_pPtr == 0){
        cudaStatus = hipMalloc((void**)&dev_p, 512 * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
    }
    else { //reuse
        dev_p = (int*)dev_pPtr;
    }

    cudaStatus = hipMalloc((void**)&dev_chances, 16 * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    if (dev_regionArrayXPtr==0){
    cudaStatus = hipMalloc((void**)&dev_regionArrayX, X_ARRAY_SIZE * sizeof(float));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        fprintf(stderr, hipGetErrorString(cudaStatus));
        goto Error;
    }
    }
    else { //reuse
        dev_regionArrayX = (float*)dev_regionArrayXPtr;
    }

    if (dev_regionArrayYPtr == 0) {
        cudaStatus = hipMalloc((void**)&dev_regionArrayY, Y_ARRAY_SIZE * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
    }
    else { //reuse
        dev_regionArrayY = (float*)dev_regionArrayYPtr;
    }

    if (dev_regionArrayZPtr == 0){
        cudaStatus = hipMalloc((void**)&dev_regionArrayZ, totalHeight * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
    }
    else { //reuse
        dev_regionArrayZ = (float*)dev_regionArrayZPtr;
    }

    if (dev_outputPtr==0)
    {
        cudaStatus = hipMalloc((void**)&dev_output, (X_ARRAY_SIZE * Y_ARRAY_SIZE * totalHeight)); //going to be bits
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipMalloc failed!");
            goto Error;
        }
    }
    else { //reuse
        dev_output = (byte*)dev_outputPtr;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_p, p, 512 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_chances, chances, 16 * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_regionArrayX, regionArrayX, X_ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_regionArrayY, regionArrayY, Y_ARRAY_SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_regionArrayZ, regionArrayZ, totalHeight * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        goto Error;
    }
    auto t_end = std::chrono::high_resolution_clock::now();
    double elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    //printf("Allocated Memory: %f\n", elapsed_time_ms);

    std::clock_t c_end = std::clock();
    double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    printf("Clocktime for inputs: %lf\n", time_elapsed_ms);

    // Launch a kernel on the GPU with one thread for each element.
    hipStream_t stream;
    hipStreamCreate(&stream);
    dim3 gridShape(X_ARRAY_SIZE, Y_ARRAY_SIZE, 1);
    generateNoise <<<gridShape, totalHeight,0, stream >>>(dev_p,dev_chances, dev_regionArrayX, dev_regionArrayY, dev_regionArrayZ, dev_output,totalHeight);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    printf("Clocktime for computation before sleep: %lf\n", time_elapsed_ms);
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    while (hipStreamQuery(stream) == hipErrorNotReady) {
        printf("Time to sleep!");
        Sleep(10);
    }

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    printf("Clocktime for computation before synchronize: %lf\n", time_elapsed_ms);


    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    printf("Clocktime for computation after synchronize: %lf\n", time_elapsed_ms);

    //save pointers for reuse
    dev_pPtr = (long long)dev_p;
    dev_regionArrayXPtr = (long long)dev_regionArrayX;
    dev_regionArrayYPtr = (long long)dev_regionArrayY;
    dev_regionArrayZPtr = (long long)dev_regionArrayZ;
    dev_outputPtr = (long long)dev_output;

    t_end = std::chrono::high_resolution_clock::now();
    elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    //printf("Finished computing: %f\n", elapsed_time_ms);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(output, dev_output, (X_ARRAY_SIZE * Y_ARRAY_SIZE * totalHeight)/8, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!", hipGetErrorString(cudaStatus));
        goto Error;
    }

    c_end = std::clock();
    time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    printf("Clocktime for after copying results: %lf\n", time_elapsed_ms);

Error:
    /*hipFree(dev_p);
    hipFree(dev_regionArrayX);
    hipFree(dev_regionArrayY);
    hipFree(dev_regionArrayZ);
    hipFree(dev_output);*/

    hipFree(dev_chances);
    
    return cudaStatus;
}


void getRegionArray(float* regionArrayX, float* regionArrayY, float* regionArrayZ, int minHeight, int maxHeight, int regionX, int regionY) {

    int minTileX = regionX * 4;
    int minTileY = regionY * 4;
    int maxTileX = minTileX + 5;
    int maxTileY = minTileY + 5;

    for (int tileX = 0; tileX < TILES_PER_REGION_AXIS; tileX++) {
        for (int x = 0; x < CHUNK_SIZE * CHUNKS_PER_TILE_AXIS; x++) {
            int worldX = (tileX+minTileX)*TILE_SIZE + x;
            regionArrayX[tileX * TILE_SIZE + x] = worldX / TINY_BLOBS;
        }
    }

    for (int tileY = 0; tileY < TILES_PER_REGION_AXIS; tileY++) {
        for (int y = 0; y < CHUNK_SIZE * CHUNKS_PER_TILE_AXIS; y++) {
            int worldY = (tileY+minTileY)*TILE_SIZE + y;
            regionArrayY[tileY * TILE_SIZE + y] = worldY / TINY_BLOBS;
        }
    }
    int totalHeight = maxHeight - minHeight;

    for (int z = 0; z < totalHeight; z++) {
        regionArrayZ[z] = (z-minHeight) / TINY_BLOBS;
    }
}

void getPArray(int* p, JNIEnv* env,jlong seed) {
    int permutation[256];

    for (int i=0; i < 256; i++) {
        permutation[i] = i;
    }

    jclass randomClass = env->FindClass("java/util/Random"); //use Java so we can keep the same seed.
    if (randomClass == NULL) {
        fprintf(stderr, "Unable to find java Random object");
        return;
    }
    jmethodID randomConstructor = env->GetMethodID(randomClass, "<init>", "(J)V");
    if (randomConstructor == NULL) {
        fprintf(stderr, "Unable to find java Random object constructor");
        return;
    }

    jmethodID nextIntMethod = env->GetMethodID(randomClass, "nextInt", "(I)I");

    jobject randomObject = env->NewObject(randomClass, randomConstructor, seed);


    for (int i = 256; i > 1; i--) {
        jint randomInt = env->CallIntMethod(randomObject, nextIntMethod, i); //random.NextInt(i)
        swap(permutation, i-1, randomInt);
    }

    for (int i = 0; i < 256; i++) {
        p[256 + i] = p[i] = permutation[i];
    }


}

void swap(int* array, int index1, int index2) {
    int temp = array[index1];
    array[index1] = array[index2];
    array[index2] = temp;
}

void getDataFromRequest(JNIEnv* env, jobject request, jlong& materialSeed, jint& regionX, jint& regionY, jint& materialMinHeight, jint& materialMaxHeight, jlong& regionXPtr, jlong& regionYPtr, jlong& regionZPtr, jlong& pPtr, jlong& outputPtr, byte*& outputArray, float*& chances) {
    jclass noiseHardwareAcceleratorRequestClass = env->FindClass("org/pepsoft/worldpainter/exporting/NoiseHardwareAcceleratorRequest");

    jmethodID getMaterialSeedMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getMaterialSeed", "()J");
    jmethodID getRegionXMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getRegionX", "()I");
    jmethodID getRegionYMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getRegionY", "()I");
    jmethodID getMaterialMinHeightMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getMaterialMinHeight", "()I");
    jmethodID getMaterialMaxHeightMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getMaterialMaxHeight", "()I");
    jmethodID getRegionXPtrMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getRegionXPtr", "()J");
    jmethodID getRegionYPtrMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getRegionYPtr", "()J");
    jmethodID getRegionZPtrMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getRegionZPtr", "()J");
    jmethodID getpPtrMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getpPtr", "()J");
    jmethodID getOutputPtrMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getOutputPtr", "()J");
    jmethodID getOutputArrayMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getOutputArray", "()Ljava/nio/ByteBuffer;");
    jmethodID getChancesMethod = env->GetMethodID(noiseHardwareAcceleratorRequestClass, "getChances", "()[F");

    materialSeed = env->CallLongMethod(request, getMaterialSeedMethod);
    regionX = env->CallIntMethod(request, getRegionXMethod);
    regionY = env->CallIntMethod(request, getRegionYMethod);
    materialMinHeight = env->CallIntMethod(request, getMaterialMinHeightMethod);
    materialMaxHeight = env->CallIntMethod(request, getMaterialMaxHeightMethod);
    regionXPtr = env->CallLongMethod(request, getRegionXPtrMethod);
    regionYPtr = env->CallLongMethod(request, getRegionYPtrMethod);
    regionZPtr = env->CallLongMethod(request, getRegionZPtrMethod);
    pPtr = env->CallLongMethod(request, getpPtrMethod);
    outputPtr = env->CallLongMethod(request, getOutputPtrMethod);
    jobject outputArrayBuffer =  env->CallObjectMethod(request, getOutputArrayMethod);
    outputArray = (byte*) env->GetDirectBufferAddress(outputArrayBuffer);
    jfloatArray chancesArray = (jfloatArray)env->CallObjectMethod(request, getChancesMethod);
    chances = env->GetFloatArrayElements(chancesArray, 0);
}


jobject createResponse(JNIEnv* env, long long dev_regionXPtr, long long dev_regionYPtr, long long dev_regionZPtr, long long dev_pPtr, long long dev_outputPtr, int totalHeight) {
    jclass noiseHardwareAcceleratorResponseClass = env->FindClass("org/pepsoft/worldpainter/exporting/NoiseHardwareAcceleratorResponse");

    jmethodID constructorMethod = env->GetMethodID(noiseHardwareAcceleratorResponseClass, "<init>", "([FJJJJJ)V");

    jlong pPtr= (jlong) dev_pPtr;
    jlong regionXPtr= (jlong) dev_regionXPtr;
    jlong regionYPtr = (jlong) dev_regionYPtr;
    jlong regionZPtr = (jlong) dev_regionZPtr;
    jlong outputPtr = (jlong) dev_outputPtr;

    jobject response = env->NewObject(noiseHardwareAcceleratorResponseClass, constructorMethod, NULL, regionXPtr, regionYPtr, regionZPtr, pPtr, outputPtr);

    return response;
}

JNIEXPORT jobject JNICALL Java_org_pepsoft_worldpainter_exporting_NoiseHardwareAccelerator_getRegionNoiseData (JNIEnv* env, jclass , jobject request) {
    auto t_start = std::chrono::high_resolution_clock::now();
    std::clock_t c_start = std::clock();

    jlong materialSeed;
    jint regionX;
    jint regionY;
    jint materialMinHeight;
    jint materialMaxHeight;
    jlong dev_regionXPtr;
    jlong dev_regionYPtr;
    jlong dev_regionZPtr;
    jlong dev_pPtr;
    jlong dev_outputPtr;
    byte* outputArray;
    float* chances;

    getDataFromRequest(env, request, materialSeed, regionX, regionY, materialMinHeight, materialMaxHeight, dev_regionXPtr, dev_regionYPtr, dev_regionZPtr, dev_pPtr, dev_outputPtr, outputArray, chances);


    const int totalHeight = materialMaxHeight - materialMinHeight;

    float regionArrayX[X_ARRAY_SIZE]; //dx
    float regionArrayY[Y_ARRAY_SIZE]; //dy
    float*  regionArrayZ; //dz but shifted
    int p[512];
    //[TILE_SIZE * TILES_PER_REGION_AXIS] [TILE_SIZE * TILES_PER_REGION_AXIS] [totalHeight]
    regionArrayZ = new float[totalHeight];
    getRegionArray(regionArrayX, regionArrayY, regionArrayZ, materialMaxHeight,materialMinHeight, regionX, regionY);

    
    getPArray(p,env,materialSeed);
    //double test = getPerlinNoiseAt(regionArrayX[128], regionArrayY[128], regionArrayZ[60 + 64],p);
    

    // Add vectors in parallel.

    hipError_t cudaStatus = noiseWithCuda(p,chances, regionArrayX, regionArrayY, regionArrayZ, outputArray,totalHeight,dev_regionXPtr, dev_regionYPtr, dev_regionZPtr, dev_pPtr, dev_outputPtr);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        delete[] regionArrayZ;
        return NULL;
    }

    std::clock_t c_end = std::clock();
    double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;
    //printf("Clocktime: %lf\n", time_elapsed_ms);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    /*cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return NULL;
    }*/
    
    jobject result = createResponse(env,dev_regionXPtr, dev_regionYPtr, dev_regionZPtr, dev_pPtr,dev_outputPtr, totalHeight);
   
    delete[] regionArrayZ;

    auto t_end = std::chrono::high_resolution_clock::now();
    auto elapsed_time_ms = std::chrono::duration<double, std::milli>(t_end - t_start).count();
    //printf("Finished in: %f Clocktime: %lf\n\n", elapsed_time_ms,time_elapsed_ms);
    return result;
}
